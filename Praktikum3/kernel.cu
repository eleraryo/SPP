#include "hip/hip_runtime.h"
//
//  kernel.cu
//
//  Created by Arya Mazaheri on 01/12/2018.
//

#include "ppm.h"
#include <algorithm>
#include <cmath>
#include <iostream>

using namespace std;

/*********** Gray Scale Filter  *********/

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__ void cuda_grayscale(int width, int height, BYTE *image,
                               BYTE *image_out) {
  // TODO: implement grayscale filter kernel

  // TODO this to cudo
  // =================================
  // for (int h = 0; h < height; ++h) {
  //   int offset_out = h * width;      // 1 color per pixel
  //   int offset     = offset_out * 3; // 3 colors per pixel

  //   for (int w = 0; w < width; ++w) {
  //     BYTE *pixel = &image[offset + w * 3];
  //     // Convert to grayscale following the "luminance" model
  //     image_out[offset_out + w] =
  //       pixel[0] * 0.2126f + // R
  //       pixel[1] * 0.7152f + // G
  //       pixel[2] * 0.0722f;  // B
  //   }
  // }
  // =================================
}

// 1D Gaussian kernel array values of a fixed size (make sure the number >
// filter size d)
// TODO: Define the cGaussian array on the constant memory

void cuda_updateGaussian(int r, double sd) {
  float fGaussian[64];
  for (int i = 0; i < 2 * r + 1; i++) {
    float x = i - r;
    fGaussian[i] = expf(-(x * x) / (2 * sd * sd));
  }
  // TODO: Copy computed fGaussian to the cGaussian on device memory
  hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), /* TODO */);
}

// TODO: implement cuda_gaussian() kernel

/*********** Bilateral Filter  *********/
// Parallel (GPU) Bilateral filter kernel
__global__ void cuda_bilateral_filter(BYTE *input, BYTE *output, int width,
                                      int height, int r, double sI, double sS) {
  // TODO: implement bilateral filter kernel
}

void gpu_pipeline(const Image &input, Image &output, int r, double sI,
                  double sS) {
  // Events to calculate gpu run time
  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // GPU related variables
  BYTE *d_input = NULL;
  BYTE *d_image_out[2] = {0}; // temporary output buffers on gpu device
  int image_size = input.cols * input.rows;
  int suggested_blockSize;   // The launch configurator returned block size
  int suggested_minGridSize; // The minimum grid size needed to achieve the
                             // maximum occupancy for a full device launch

  // ******* Grayscale kernel launch *************

  // Creating the block size for grayscaling kernel
  hipOccupancyMaxPotentialBlockSize(&suggested_minGridSize,
                                     &suggested_blockSize, cuda_grayscale);

  int block_dim_x, block_dim_y;

  dim3 gray_block(/* TODO */);

  // TODO: Calculate grid size to cover the whole image

  // Allocate the intermediate image buffers for each step
  Image img_out(input.cols, input.rows, 1, "P5");
  for (int i = 0; i < 2; i++) {
    // TODO: allocate memory on the device
    hipMalloc(&d_input, image_size);
    // TODO: intialize allocated memory on device to zero
    hipMemcpy(&d_input, &input, image_size, hipMemcpyHostToDevice);
  }

  // copy input image to device
  // TODO: Allocate memory on device for input image
  // TODO: Copy input image into the device memory

  hipEventRecord(start, 0); // start timer
  // Convert input image to grayscale
  // TODO: Launch cuda_grayscale()
  hipEventRecord(stop, 0); // stop timer
  hipEventSynchronize(stop);

  // Calculate and print kernel run time
  hipEventElapsedTime(&time, start, stop);
  cout << "GPU Grayscaling time: " << time << " (ms)\n";
  cout << "Launched blocks of size " << gray_block.x * gray_block.y << endl;

  // TODO: transfer image from device to the main memory for saving onto the
  // disk
  savePPM(img_out, "image_gpu_gray.ppm");

  // ******* Bilateral filter kernel launch *************

  // Creating the block size for grayscaling kernel
  hipOccupancyMaxPotentialBlockSize(
      &suggested_minGridSize, &suggested_blockSize, cuda_bilateral_filter);

  dim3 bilateral_block(/* TODO */);

  // TODO: Calculate grid size to cover the whole image

  // Create gaussain 1d array
  cuda_updateGaussian(r, sS);

  hipEventRecord(start, 0); // start timer
  // TODO: Launch cuda_bilateral_filter()
  hipEventRecord(stop, 0); // stop timer
  hipEventSynchronize(stop);

  // Calculate and print kernel run time
  hipEventElapsedTime(&time, start, stop);
  cout << "GPU Bilateral Filter time: " << time << " (ms)\n";
  cout << "Launched blocks of size " << bilateral_block.x * bilateral_block.y
       << endl;

  // Copy output from device to host
  // TODO: transfer image from device to the main memory for saving onto the
  // disk

  // ************** Finalization, cleaning up ************

  // Free GPU variables
  // TODO: Free device allocated memory
}
