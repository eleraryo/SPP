#include "hip/hip_runtime.h"
//
//  kernel.cu
//
//  Created by Arya Mazaheri on 01/12/2018.
//

#include "ppm.h"
#include <algorithm>
#include <cmath>
#include <iostream>

using namespace std;
/*********** Gray Scale Filter  *********/

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__ void cuda_grayscale(int width, int height, BYTE *image,
                               BYTE *image_out) {
  // DONE: implement grayscale filter kernel
  int w = blockIdx.x * blockDim.x + threadIdx.x;
  int h = blockIdx.y * blockDim.y + threadIdx.y;

  if (w >= width || h >= height)
    return;
  int position = h * width + w;
  BYTE *pixel = &image[position * 3];
  image_out[position] = pixel[0] * 0.2126f + // R
                        pixel[1] * 0.7152f + // G
                        pixel[2] * 0.0722f;  // B
  return;
}

// 1D Gaussian kernel array values of a fixed size (make sure the number >
// filter size d)
// DONE: Define the cGaussian array on the constant memory
__constant__ float cGaussian[64];

void cuda_updateGaussian(int r, double sd) {
  float fGaussian[64];
  for (int i = 0; i < 2 * r + 1; i++) {
    float x = i - r;
    fGaussian[i] = expf(-(x * x) / (2 * sd * sd));
  }
  // DONE: Copy computed fGaussian to the cGaussian on device memory
  auto status =
      hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, sizeof(float) * (2 * r + 1));
  // cout << "update gaussian memcpy: " << hipGetErrorName(status) << endl;
}

// DONE: implement cuda_gaussian() kernel
// Gaussian function for range difference
__device__ double cuda_gaussian(float x, double sigma) {
  return expf(-(powf(x, 2)) / (2 * powf(sigma, 2)));
}

/*********** Bilateral Filter  *********/
// Parallel (GPU) Bilateral filter kernel
__global__ void cuda_bilateral_filter(BYTE *input, BYTE *output, int width,
                                      int height, int r, double sI, double sS) {
  // DONE: implement bilateral filter kernel
  //
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) {
    return;
  }

  double iFiltered = 0;
  double wP = 0;
  unsigned char centerPx = input[y * width + x];
  for (int dy = -r; dy <= r; ++dy) {
    int neighborY = y + dy;
    if (neighborY < 0)
      neighborY = 0;
    else if (neighborY >= height)
      neighborY = height - 1;
    for (int dx = -r; dx <= r; ++dx) {
      int neighborX = x + dx;
      if (neighborX < 0)
        neighborX = 0;
      else if (neighborX >= width)
        neighborX = width - 1;
      unsigned char currPx = input[neighborY * width + neighborX];

      double wG = (cGaussian[dy + r] * cGaussian[dx + r]) *
                  cuda_gaussian(centerPx - currPx, sI);
      iFiltered += wG * currPx;
      wP += wG;
    }
  }
  output[y * width + x] = iFiltered / wP;
}

void gpu_pipeline(const Image &input, Image &output, int r, double sI,
                  double sS) {
  // Events to calculate gpu run time
  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // GPU related variables
  BYTE *d_input = NULL;
  BYTE *d_image_out = NULL; // temporary output buffers on gpu device
  int image_size = input.cols * input.rows;
  int suggested_blockSize;   // The launch configurator returned block size
  int suggested_minGridSize; // The minimum grid size needed to achieve
                             // the maximum occupancy for a full device
                             // launch

  // ******* Grayscale kernel launch *************

  // Creating the block size for grayscaling kernel
  hipOccupancyMaxPotentialBlockSize(&suggested_minGridSize,
                                     &suggested_blockSize, cuda_grayscale);

  int block_dim_x = sqrt(suggested_blockSize);
  int block_dim_y = sqrt(suggested_blockSize);

  // DONE: Calculate grid size to cover the whole image
  dim3 threadsPerBlock(block_dim_x, block_dim_y); // this was grey_block
  dim3 numBlocks((input.cols + block_dim_x - 1) / block_dim_x,
                 (input.rows + block_dim_y - 1) / block_dim_y);

  auto out = hipMalloc(&d_image_out, image_size * sizeof(BYTE));
  // cout << hipGetErrorName(out) << " malloc did this " << endl;
  // DONE: intialize allocated memory on device to zero
  auto memset_err = hipMemset(d_image_out, 0, image_size * sizeof(BYTE));
  // cout << hipGetErrorName(memset_err) << " memset did this " << endl;
  // hipMemcpy(&d_image_out, &output.pixels, image_size * sizeof(BYTE),
  //            hipMemcpyHostToDevice);

  // copy input image to device
  // DONE: Allocate memory on device for input image
  auto out2 = hipMalloc(&d_input, image_size * 3 * sizeof(BYTE));
  // cout << hipGetErrorName(out2) << " malloc did this " << endl;
  // DONE: Copy input image into the device memory
  auto memcpy_err =
      hipMemcpy(d_input, input.pixels, image_size * 3 * sizeof(BYTE),
                 hipMemcpyHostToDevice);
  // cout << hipGetErrorName(memcpy_err) << " memcopy to device did this "
  //      << endl;

  hipEventRecord(start, 0); // start timer
  // Convert input image to grayscale

  // DONE: Launch cuda_grayscale()
  printf("call cuda_greyscale with : {%d, %d} threads per block and : "
         "{%d, %d} "
         "number of blocks \n",
         threadsPerBlock.x, threadsPerBlock.y, numBlocks.x, numBlocks.y);
  cuda_grayscale<<<threadsPerBlock, numBlocks>>>(input.cols, input.rows,
                                                 d_input, d_image_out);

  hipEventRecord(stop, 0); // stop timer
  hipEventSynchronize(stop);

  // Calculate and print kernel run time
  hipEventElapsedTime(&time, start, stop);
  cout << "GPU Grayscaling time: " << time << " (ms)\n";

  // DONE: transfer image from device to the main memory for saving onto
  // the disk
  auto memcpy_err2 =
      hipMemcpy(output.pixels, d_image_out, image_size * sizeof(BYTE),
                 hipMemcpyDeviceToHost);
  // cout << hipGetErrorName(memcpy_err2) << " memcopy to host did this " <<
  // endl;
  savePPM(output, "image_gpu_gray.ppm");

  // ******* Bilateral filter kernel launch *************

  // Creating the block size for grayscaling kernel
  hipOccupancyMaxPotentialBlockSize(
      &suggested_minGridSize, &suggested_blockSize, cuda_bilateral_filter);

  block_dim_x = sqrt(suggested_blockSize);
  block_dim_y = sqrt(suggested_blockSize);

  // DONE: Calculate grid size to cover the whole image
  dim3 threadsPerBlockBiliteral(block_dim_x,
                                block_dim_y); // this was biliteral_block
  // DONE: Calculate grid size to cover the whole image
  dim3 numBlocksBiliteral((input.cols + block_dim_x - 1) / block_dim_x,
                          (input.rows + block_dim_y - 1) / block_dim_y);

  // Create gaussain 1d array
  cuda_updateGaussian(r, sS);

  BYTE *d_bil_image_out = NULL; // temporary output buffers on gpu device
  // create zeroes image output for biliteral
  out = hipMalloc(&d_bil_image_out, image_size * sizeof(BYTE));
  // cout << hipGetErrorName(out) << " malloc biliteral did this " << endl;
  // DONE: intialize allocated memory on device to zero
  memset_err = hipMemset(d_bil_image_out, 0, image_size * sizeof(BYTE));
  // cout << hipGetErrorName(memset_err) << " memset biliteral did this " <<
  // endl;

  hipEventRecord(start, 0); // start timer
  // DONE: Launch cuda_bilateral_filter()
  cuda_bilateral_filter<<<threadsPerBlockBiliteral, numBlocksBiliteral>>>(
      d_image_out, d_bil_image_out, input.cols, input.rows, r, sI, sS);
  hipEventRecord(stop, 0); // stop timer
  hipEventSynchronize(stop);

  // Calculate and print kernel run time
  hipEventElapsedTime(&time, start, stop);
  cout << "GPU Bilateral Filter time: " << time << " (ms)\n";

  // Copy output from device to host
  // DONE: transfer image from device to the main memory for saving onto
  // the disk

  out = hipMemcpy(output.pixels, d_bil_image_out, image_size * sizeof(BYTE),
                   hipMemcpyDeviceToHost);
  // cout << hipGetErrorName(out) << " memcopy biliteral to host did this "
  //      << endl;
  savePPM(output, "image_bil_gpu_gray.ppm");
  // ************** Finalization, cleaning up ************

  // Free GPU variables
  // DONE: Free device allocated memory
  out = hipFree(d_input);
  // cout << hipGetErrorName(out) << " free d_input did this " << endl;
  out = hipFree(d_image_out);
  // cout << hipGetErrorName(out) << " free d_image_out did this " << endl;
  out = hipFree(d_bil_image_out);
  // cout << hipGetErrorName(out) << " free d_bil_image_out did this " << endl;
}
