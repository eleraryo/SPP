#include "hip/hip_runtime.h"
//
//  kernel.cu
//
//  Created by Arya Mazaheri on 01/12/2018.
//

#include "ppm.h"
#include <algorithm>
#include <cmath>
#include <iostream>

using namespace std;

/*********** Gray Scale Filter  *********/

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__ void cuda_grayscale(int width, int height, BYTE *image,
                               BYTE *image_out) {
  // TODO: implement grayscale filter kernel
  // iterate over Matrix style element
  int w = blockIdx.x * blockDim.x + threadIdx.x;
  int h = blockIdx.y * blockDim.y + threadIdx.y;
  // int w = threadIdx.x;
  // int h = threadIdx.y;

  if (w < width && h < height) {
    printf("trying to write pixel %d, %d\n", w, h);
    // BYTE *pixel = &image[w];
    image_out[w * h + w] = 254;
    // int offset_out = h * width;  // 1 color per pixel
    // int offset = offset_out * 3; // 3 colors per pixel
    // BYTE *pixel = &image[offset + w * 3];
    // // Convert to grayscale following the "luminance" model
    // image_out[offset_out + w] = pixel[0] * 0.2126f + // R
    //                             pixel[1] * 0.7152f + // G
    //                             pixel[2] * 0.0722f;  // B
    return;
  } else
    return;
}

// 1D Gaussian kernel array values of a fixed size (make sure the number >
// filter size d)
// DONE: Define the cGaussian array on the constant memory
__constant__ float cGaussian[64];

void cuda_updateGaussian(int r, double sd) {
  float fGaussian[64];
  for (int i = 0; i < 2 * r + 1; i++) {
    float x = i - r;
    fGaussian[i] = expf(-(x * x) / (2 * sd * sd));
  }
  // DONE: Copy computed fGaussian to the cGaussian on device memory
  hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, sizeof(float) * (2 * r + 1));
}

// TODO: implement cuda_gaussian() kernel

/*********** Bilateral Filter  *********/
// Parallel (GPU) Bilateral filter kernel
__global__ void cuda_bilateral_filter(BYTE *input, BYTE *output, int width,
                                      int height, int r, double sI, double sS) {
  // TODO: implement bilateral filter kernel
  //
  // __global__ void d_bilateral_filter(uint * od, int w, int h, float e_d, int
  // r,
  //                                    hipTextureObject_t rgbaTex) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // if (x >= w || y >= h) {
  //   return;
  // }

  // float sum = 0.0f;
  // float factor;
  // float4 t = {0.f, 0.f, 0.f, 0.f};
  // float4 center = tex2D<float4>(rgbaTex, x, y);

  // for (int i = -r; i <= r; i++) {
  //   for (int j = -r; j <= r; j++) {
  //     float4 curPix = tex2D<float4>(rgbaTex, x + j, y + i);
  //     factor = cGaussian[i + r] * cGaussian[j + r] * // domain factor
  //              euclideanLen(curPix, center, e_d);    // range factor

  //     t += factor * curPix;
  //     sum += factor;
  //   }
  // }

  // od[y * w + x] = rgbaFloatToInt(t / sum);
}

void gpu_pipeline(const Image &input, Image &output, int r, double sI,
                  double sS) {
  // Events to calculate gpu run time
  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // GPU related variables
  BYTE *d_input = NULL;
  // BYTE *d_image_out[2] = {0}; // temporary output buffers on gpu device //
  // TODO check why this was {0} instead of NULL
  BYTE *d_image_out = NULL; // temporary output buffers on gpu device
  int image_size = input.cols * input.rows;
  int suggested_blockSize;   // The launch configurator returned block size
  int suggested_minGridSize; // The minimum grid size needed to achieve the
                             // maximum occupancy for a full device launch

  // ******* Grayscale kernel launch *************

  // Creating the block size for grayscaling kernel
  hipOccupancyMaxPotentialBlockSize(&suggested_minGridSize,
                                     &suggested_blockSize, cuda_grayscale);

  int block_dim_x = sqrt(suggested_blockSize);
  int block_dim_y = sqrt(suggested_blockSize);
  cout << "suggested gridsize:" << suggested_minGridSize
       << "and suggested blocksize: " << suggested_blockSize << endl;
  cout << "chosen block_dim " << block_dim_x << endl;

  // dim3 gray_block((suggested_minGridSize + block_dim_x - 1) / block_dim_x,
  //                 (suggested_minGridSize + block_dim_y - 1) / block_dim_y);
  // dim3 gray_block(block_dim_x, block_dim_y);
  dim3 gray_block(32, 32);

  // DONE: Calculate grid size to cover the whole image
  // dim3 gridSize((input.cols + block_dim_x - 1) / block_dim_x,
  //               (input.rows + block_dim_y - 1) / block_dim_y);
  dim3 gridSize(suggested_minGridSize);

  // Allocate the intermediate image buffers for each step
  Image img_out(input.cols, input.rows, 1, "P5");
  for (int i = 0; i < 2; i++) {
    // DONE: allocate memory on the device
    hipMalloc(&d_image_out, image_size);
    // DONE: intialize allocated memory on device to zero
    hipMemset(&d_image_out, 0, image_size);
  }

  // copy input image to device
  // DONE: Allocate memory on device for input image
  hipMalloc(&d_input, image_size);
  // DONE: Copy input image into the device memory
  hipMemcpy(&d_input, &input, image_size, hipMemcpyHostToDevice);

  hipEventRecord(start, 0); // start timer
  // Convert input image to grayscale

  // DONE: Launch cuda_grayscale()
  printf(
      "call cuda_greyscale with gridsize: {%d, %d} and blocksize: {%d, %d} \n",
      gridSize, gray_block);
  cuda_grayscale<<<gridSize, gray_block>>>(input.cols, input.rows, &d_input,
                                           &d_image_out);
  printf("finished cuda_greyscale\n");

  hipEventRecord(stop, 0); // stop timer
  hipEventSynchronize(stop);

  // Calculate and print kernel run time
  hipEventElapsedTime(&time, start, stop);
  cout << "GPU Grayscaling time: " << time << " (ms)\n";
  // TODO back in: cout << "Launched blocks of size " << gray_block.x *
  // gray_block.y << endl;

  // DONE: transfer image from device to the main memory for saving onto the
  // disk
  hipMemcpy(&d_image_out, &input, image_size, hipMemcpyDeviceToHost);
  savePPM(img_out, "image_gpu_gray.ppm");

  // ******* Bilateral filter kernel launch *************

  // Creating the block size for grayscaling kernel
  hipOccupancyMaxPotentialBlockSize(
      &suggested_minGridSize, &suggested_blockSize, cuda_bilateral_filter);

  dim3 bilateral_block(/* TODO */);

  // TODO: Calculate grid size to cover the whole image

  // Create gaussain 1d array
  cuda_updateGaussian(r, sS);

  hipEventRecord(start, 0); // start timer
  // TODO: Launch cuda_bilateral_filter()
  hipEventRecord(stop, 0); // stop timer
  hipEventSynchronize(stop);

  // Calculate and print kernel run time
  hipEventElapsedTime(&time, start, stop);
  cout << "GPU Bilateral Filter time: " << time << " (ms)\n";
  // TODO back in: cout << "Launched blocks of size " << bilateral_block.x *
  // bilateral_block.y << endl;

  // Copy output from device to host
  // TODO: transfer image from device to the main memory for saving onto the
  // disk

  // ************** Finalization, cleaning up ************

  // Free GPU variables
  // DONE: Free device allocated memory
  hipFree(&d_input);
  hipFree(&d_image_out);
}
