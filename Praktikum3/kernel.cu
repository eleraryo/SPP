#include "hip/hip_runtime.h"
//
//  kernel.cu
//
//  Created by Arya Mazaheri on 01/12/2018.
//

#include "ppm.h"
#include <algorithm>
#include <cmath>
#include <iostream>

using namespace std;

/*********** Gray Scale Filter  *********/

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__ void cuda_grayscale(int width, int height, BYTE *image,
                               BYTE *image_out) {
  // TODO: implement grayscale filter kernel
  // iterate over Matrix style element
  int w = blockIdx.x * blockDim.x + threadIdx.x;
  int h = blockIdx.y * blockDim.y + threadIdx.y;

  if (w >= width || h >= height)
    return;
  // printf("trying to write pixel %d, %d\n", w, h);
  // TODO something is off with position
  int position = h * width + w;
  BYTE *pixel = &image[position * 3];
  // TODO fix only reading 0 as pixel values
  // printf("pixel values are %u, %u, %u\n", pixel[0], pixel[1], pixel[2]);
  image_out[position] = pixel[0] * 0.2126f + // R
                        pixel[1] * 0.7152f + // G
                        pixel[2] * 0.0722f;  // B
  // image_out[position] = 254;
  // printf("new pixel value is %u @ %d \n", image_out[position], position);
  return;
}

// 1D Gaussian kernel array values of a fixed size (make sure the number >
// filter size d)
// DONE: Define the cGaussian array on the constant memory
__constant__ float cGaussian[64];

void cuda_updateGaussian(int r, double sd) {
  float fGaussian[64];
  for (int i = 0; i < 2 * r + 1; i++) {
    float x = i - r;
    fGaussian[i] = expf(-(x * x) / (2 * sd * sd));
  }
  // DONE: Copy computed fGaussian to the cGaussian on device memory
  hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, sizeof(float) * (2 * r + 1));
}

// TODO: implement cuda_gaussian() kernel

/*********** Bilateral Filter  *********/
// Parallel (GPU) Bilateral filter kernel
__global__ void cuda_bilateral_filter(BYTE *input, BYTE *output, int width,
                                      int height, int r, double sI, double sS) {
  // TODO: implement bilateral filter kernel
  //
  // __global__ void d_bilateral_filter(uint * od, int w, int h, float e_d, int
  // r,
  //                                    hipTextureObject_t rgbaTex) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // if (x >= w || y >= h) {
  //   return;
  // }

  // float sum = 0.0f;
  // float factor;
  // float4 t = {0.f, 0.f, 0.f, 0.f};
  // float4 center = tex2D<float4>(rgbaTex, x, y);

  // for (int i = -r; i <= r; i++) {
  //   for (int j = -r; j <= r; j++) {
  //     float4 curPix = tex2D<float4>(rgbaTex, x + j, y + i);
  //     factor = cGaussian[i + r] * cGaussian[j + r] * // domain factor
  //              euclideanLen(curPix, center, e_d);    // range factor

  //     t += factor * curPix;
  //     sum += factor;
  //   }
  // }

  // od[y * w + x] = rgbaFloatToInt(t / sum);
}

void gpu_pipeline(const Image &input, Image &output, int r, double sI,
                  double sS) {
  // Events to calculate gpu run time
  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // GPU related variables
  BYTE *d_input = NULL;
  // BYTE *d_image_out[2] = {0}; // temporary output buffers on gpu device //
  // TODO check why this was {0} instead of NULL
  BYTE *d_image_out = NULL; // temporary output buffers on gpu device
  int image_size = input.cols * input.rows;
  int suggested_blockSize;   // The launch configurator returned block size
  int suggested_minGridSize; // The minimum grid size needed to achieve the
                             // maximum occupancy for a full device launch

  // ******* Grayscale kernel launch *************

  // Creating the block size for grayscaling kernel
  hipOccupancyMaxPotentialBlockSize(&suggested_minGridSize,
                                     &suggested_blockSize, cuda_grayscale);

  int block_dim_x = sqrt(suggested_blockSize);
  int block_dim_y = sqrt(suggested_blockSize);

  // DONE: Calculate grid size to cover the whole image
  dim3 threadsPerBlock(block_dim_x, block_dim_y); // this was grey_block
  dim3 numBlocks((input.cols + block_dim_x - 1) / block_dim_x,
                 (input.rows + block_dim_y - 1) / block_dim_y);

  // Allocate the intermediate image buffers for each step
  // Image img_out(input.cols, input.rows, 1, "P5");
  // for (int i = 0; i < 2; i++) {
  //   // TODO Why do we do this twice but don't use the second one?
  //   // DONE: allocate memory on the device
  //   hipMalloc(&d_image_out, image_size * sizeof(BYTE *));
  //   // DONE: intialize allocated memory on device to zero
  //   hipMemset(&d_image_out, 0, image_size * sizeof(BYTE *));
  // }
  auto out = hipMalloc(&d_image_out, image_size * sizeof(BYTE));
  cout << hipGetErrorName(out) << " malloc did this " << endl;
  // DONE: intialize allocated memory on device to zero
  auto memset_err = hipMemset(d_image_out, 0, image_size * sizeof(BYTE));
  cout << hipGetErrorName(memset_err) << " memset did this " << endl;
  // hipMemcpy(&d_image_out, &output.pixels, image_size * sizeof(BYTE),
  //            hipMemcpyHostToDevice);

  // copy input image to device
  // DONE: Allocate memory on device for input image
  auto out2 = hipMalloc(&d_input, image_size * 3 * sizeof(BYTE));
  cout << hipGetErrorName(out2) << " malloc did this " << endl;
  // DONE: Copy input image into the device memory
  auto memcpy_err =
      hipMemcpy(d_input, input.pixels, image_size * 3 * sizeof(BYTE),
                 hipMemcpyHostToDevice);
  cout << hipGetErrorName(memcpy_err) << " memcopy to device did this "
       << endl;

  hipEventRecord(start, 0); // start timer
  // Convert input image to grayscale

  // DONE: Launch cuda_grayscale()
  printf("call cuda_greyscale with : {%d, %d} threads per block and : {%d, %d} "
         "number of blocks \n",
         threadsPerBlock.x, threadsPerBlock.y, numBlocks.x, numBlocks.y);
  cuda_grayscale<<<threadsPerBlock, numBlocks>>>(input.cols, input.rows,
                                                 d_input, d_image_out);
  printf("finished cuda_greyscale\n");

  hipEventRecord(stop, 0); // stop timer
  hipEventSynchronize(stop);

  // Calculate and print kernel run time
  hipEventElapsedTime(&time, start, stop);
  cout << "GPU Grayscaling time: " << time << " (ms)\n";
  // TODO back in: cout << "Launched blocks of size " << gray_block.x *
  // gray_block.y << endl;

  // DONE: transfer image from device to the main memory for saving onto the
  // disk
  auto memcpy_err2 =
      hipMemcpy(output.pixels, d_image_out, image_size * sizeof(BYTE),
                 hipMemcpyDeviceToHost);
  cout << hipGetErrorName(memcpy_err2) << " memcopy to host did this " << endl;
  savePPM(output, "image_gpu_gray.ppm");

  // ******* Bilateral filter kernel launch *************

  // Creating the block size for grayscaling kernel
  hipOccupancyMaxPotentialBlockSize(
      &suggested_minGridSize, &suggested_blockSize, cuda_bilateral_filter);

  dim3 bilateral_block(/* TODO */);

  // TODO: Calculate grid size to cover the whole image

  // Create gaussain 1d array
  cuda_updateGaussian(r, sS);

  hipEventRecord(start, 0); // start timer
  // TODO: Launch cuda_bilateral_filter()
  hipEventRecord(stop, 0); // stop timer
  hipEventSynchronize(stop);

  // Calculate and print kernel run time
  hipEventElapsedTime(&time, start, stop);
  cout << "GPU Bilateral Filter time: " << time << " (ms)\n";
  // TODO back in: cout << "Launched blocks of size " << bilateral_block.x *
  // bilateral_block.y << endl;

  // Copy output from device to host
  // TODO: transfer image from device to the main memory for saving onto the
  // disk

  // ************** Finalization, cleaning up ************

  // Free GPU variables
  // DONE: Free device allocated memory
  hipFree(&d_input);
  hipFree(&d_image_out);
}
