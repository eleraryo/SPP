#include "hip/hip_runtime.h"
//
//  kernel.cu
//
//  Created by Arya Mazaheri on 01/12/2018.
//

#include <iostream>
#include <algorithm>
#include <cmath>
#include "ppm.h"

using namespace std;

/*********** Gray Scale Filter  *********/

/**
 * Converts a given 24bpp image into 8bpp grayscale using the GPU.
 */
__global__
void cuda_grayscale(int width, int height, BYTE *image, BYTE *image_out)
{
    //TODO: implement grayscale filter kernel
}


// 1D Gaussian kernel array values of a fixed size (make sure the number > filter size d)
//TODO: Define the cGaussian array on the constant memory 

void cuda_updateGaussian(int r, double sd)
{
	float fGaussian[64];
	for (int i = 0; i < 2*r +1 ; i++)
	{
		float x = i - r;
		fGaussian[i] = expf(-(x*x) / (2 * sd*sd));
	}
	//TODO: Copy computed fGaussian to the cGaussian on device memory 
	hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), /* TODO */);
}

//TODO: implement cuda_gaussian() kernel


/*********** Bilateral Filter  *********/
// Parallel (GPU) Bilateral filter kernel
__global__ void cuda_bilateral_filter(BYTE* input, BYTE* output,
	int width, int height,
	int r, double sI, double sS)
{
	//TODO: implement bilateral filter kernel 
}


void gpu_pipeline(const Image & input, Image & output, int r, double sI, double sS)
{
	// Events to calculate gpu run time
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// GPU related variables
	BYTE *d_input = NULL;
	BYTE *d_image_out[2] = {0}; //temporary output buffers on gpu device
	int image_size = input.cols*input.rows;
	int suggested_blockSize;   // The launch configurator returned block size 
	int suggested_minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch

	// ******* Grayscale kernel launch *************

	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( /* TODO */ );
        
	int block_dim_x, block_dim_y; 

	dim3 gray_block(/* TODO */);

	//TODO: Calculate grid size to cover the whole image

	// Allocate the intermediate image buffers for each step
	Image img_out(input.cols, input.rows, 1, "P5");
	for (int i = 0; i < 2; i++)
	{  
		//TODO: allocate memory on the device
		//TODO: intialize allocated memory on device to zero 
	}

	//copy input image to device
	//TODO: Allocate memory on device for input image 
	//TODO: Copy input image into the device memory

	hipEventRecord(start, 0); // start timer
	// Convert input image to grayscale
	//TODO: Launch cuda_grayscale() 
	hipEventRecord(stop, 0); // stop timer
	hipEventSynchronize(stop);

	// Calculate and print kernel run time
	hipEventElapsedTime(&time, start, stop);
	cout << "GPU Grayscaling time: " << time << " (ms)\n";
	cout << "Launched blocks of size " << gray_block.x * gray_block.y << endl;
    
	//TODO: transfer image from device to the main memory for saving onto the disk 
	savePPM(img_out, "image_gpu_gray.ppm");
	

	// ******* Bilateral filter kernel launch *************
	
	//Creating the block size for grayscaling kernel
	hipOccupancyMaxPotentialBlockSize( /* TODO */ ); 

	dim3 bilateral_block(/* TODO */);

	//TODO: Calculate grid size to cover the whole image

	// Create gaussain 1d array
	cuda_updateGaussian(r,sS);

	hipEventRecord(start, 0); // start timer
	//TODO: Launch cuda_bilateral_filter() 
	hipEventRecord(stop, 0); // stop timer
	hipEventSynchronize(stop);

	// Calculate and print kernel run time
	hipEventElapsedTime(&time, start, stop);
	cout << "GPU Bilateral Filter time: " << time << " (ms)\n";
	cout << "Launched blocks of size " << bilateral_block.x * bilateral_block.y << endl;

	// Copy output from device to host
	//TODO: transfer image from device to the main memory for saving onto the disk 


	// ************** Finalization, cleaning up ************

	// Free GPU variables
	//TODO: Free device allocated memory 
}
